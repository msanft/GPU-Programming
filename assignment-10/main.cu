#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdint>
#include <ctime>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

#define TILE_WIDTH 32
#define MATRIX_SIZE 5000 // square
#define EPSILON 0.1f
#define STREAM_COUNT 2

// <--- HELPER FUNCTIONS --->

/*
populate_matrix fills the matrix `res` with random float values.

Values are in the range [0, 25.5] (25.5 due to 0xff=255/10) to avoid precision
issues in comparison.

See https://docs.nvidia.com/cuda/floating-point/index.html.

The caller is expected to seed the RNG.
*/
void populate_matrix(float_t *res, uint32_t width, uint32_t height) {
  for (uint32_t i = 0; i < height; i++)
    for (uint32_t j = 0; j < width; j++)
      // avoid precision issues in comparison by giving an upper bound
      res[i * width + j] = (float_t)(rand() & 0xFF) / 10.0f;
}

/*
get_time_diff_ns calculates the difference between two timespec structs in
nanoseconds.
*/
int64_t get_time_diff_ns(struct timespec start, struct timespec end) {
  int64_t diff = (int64_t)(end.tv_sec - start.tv_sec) * (int64_t)1e9 +
                 (int64_t)(end.tv_nsec - start.tv_nsec);
  return diff;
}

// <--- CPU FUNCTIONS --->

/*
_multiply_matrices_cpu performs matrix multiplication on `m1` and `m2` and
stores the result in `res`.

It should not be used directly; instead, use the wrapper function
`multiply_matrices_cpu`.
*/
void _multiply_matrices_cpu(float_t *res, float_t *m1, float_t *m2,
                            uint32_t result_width, uint32_t result_height,
                            uint32_t shared_dim) {
  for (uint32_t i = 0; i < result_height; i++) {
    for (uint32_t j = 0; j < result_width; j++) {
      float_t p_value = 0;

      for (uint32_t k = 0; k < shared_dim; k++)
        p_value += m1[i * shared_dim + k] * m2[k * result_width + j];

      res[i * result_width + j] = p_value;
    }
  }
}

/*
multiply_matrices_cpu performs matrix multiplication on `m1` and `m2` using the
local tiled matrix multiplication algorithm and stores the result in `res`.

It also times the operation and returns the time taken to nanosecond-precision.
*/
int64_t multiply_matrices_cpu(float_t *res, float_t *m1, float_t *m2,
                              uint32_t result_width, uint32_t result_height,
                              uint32_t shared_dim) {
  struct timespec start, end;

  clock_gettime(CLOCK_MONOTONIC, &start);

  _multiply_matrices_cpu(res, m1, m2, result_width, result_height, shared_dim);

  clock_gettime(CLOCK_MONOTONIC, &end);

  return get_time_diff_ns(start, end);
}

/*
compare_outputs checks if the matrices in `res_1` and `res_2`
match by comparing each element with the `epsilon` error margin.

See https://docs.nvidia.com/cuda/floating-point/index.html.

If a mismatch is found, it will be printed to stderr and 1 will be returned.
Otherwise, 0 will be returned.
*/
uint32_t compare_outputs(float_t *res_1, float_t *res_2, float_t epsilon,
                         uint32_t width, uint32_t height) {
  for (uint32_t y = 0; y < height; y++)
    for (uint32_t x = 0; x < width; x++)
      if ((fabs(res_1[y * width + x] - res_2[y * width + x]) /
           fabs(res_1[y * width + x])) > epsilon) {
        fprintf(stderr, "Results differ significantly at index (%d, %d).\n", x,
                y);
        fprintf(stderr, "Result 1: %f, Result 2: %f\n", res_1[y * width + x],
                res_2[y * width + x]);
        fprintf(stderr, "Relative difference: %f\n",
                fabs(res_1[y * width + x] - res_2[y * width + x]) /
                    fabs(res_1[y * width + x]));
        return 1;
      }

  return 0;
}

// <--- GPU FUNCTIONS --->

/*
_multiply_matrices_gpu performs matrix multiplication on `m1` and `m2` using the
local tiled matrix multiplication algorithm and stores the result in `res`.

It should not be used directly; instead, use the wrapper function
`multiply_matrices_gpu`.
*/
__global__ void _multiply_matrices_gpu(float_t *res, float_t *m1, float_t *m2,
                                       uint32_t result_width,
                                       uint32_t result_height,
                                       uint32_t shared_dim) {
  uint32_t row = blockIdx.y * TILE_WIDTH + threadIdx.y;
  uint32_t col = blockIdx.x * TILE_WIDTH + threadIdx.x;

  float_t p_value = 0;

  for (int k = 0; k < shared_dim; k++)
    p_value += m1[row * shared_dim + k] * m2[k * result_width + col];

  // bounds check required due to round-up in grid size
  if (row < result_height && col < result_width)
    res[row * result_width + col] = p_value;
}

/*
multiply_matrices_gpu_multistream performs matrix multiplication on `m1` and
`m2` and stores the result in `res`.

It uses multiple streams.

It also performs CUDA memory management and kernel invocation, using the
specified `block_width` and `block_height` for the kernel, as well as timing the
operation.

It returns the time taken to nanosecond-precision.
*/
int64_t multiply_matrices_gpu_multistream(float_t *res, float_t *m1,
                                          float_t *m2) {
  // Create compartments
  float_t *h_half_1 =
      (float_t *)malloc(MATRIX_SIZE * (MATRIX_SIZE / 2) * sizeof(float_t));
  float_t *h_half_2 =
      (float_t *)malloc(MATRIX_SIZE * (MATRIX_SIZE / 2) * sizeof(float_t));
  float_t *v_half_1 =
      (float_t *)malloc(MATRIX_SIZE * (MATRIX_SIZE / 2) * sizeof(float_t));
  float_t *v_half_2 =
      (float_t *)malloc(MATRIX_SIZE * (MATRIX_SIZE / 2) * sizeof(float_t));

  for (uint32_t i = 0; i < (MATRIX_SIZE / 2); i++) {
    memcpy(h_half_1 + (i * MATRIX_SIZE), m1 + (i * MATRIX_SIZE),
           MATRIX_SIZE * sizeof(float_t));
    memcpy(h_half_2 + (i * MATRIX_SIZE),
           m1 + ((MATRIX_SIZE / 2) * MATRIX_SIZE) + (i * MATRIX_SIZE),
           MATRIX_SIZE * sizeof(float_t));
  }

  for (uint32_t i = 0; i < (MATRIX_SIZE / 2); i++) {
    memcpy(v_half_1 + (i * (MATRIX_SIZE / 2)), m2 + (i * MATRIX_SIZE),
           (MATRIX_SIZE / 2) * sizeof(float_t));
    memcpy(v_half_2 + (i * (MATRIX_SIZE / 2)),
           m2 + (MATRIX_SIZE / 2) + (i * MATRIX_SIZE),
           (MATRIX_SIZE / 2) * sizeof(float_t));
  }

  float_t *upper_left = (float_t *)malloc((MATRIX_SIZE / 2) *
                                          (MATRIX_SIZE / 2) * sizeof(float_t));
  float_t *upper_right = (float_t *)malloc((MATRIX_SIZE / 2) *
                                           (MATRIX_SIZE / 2) * sizeof(float_t));
  float_t *lower_left = (float_t *)malloc((MATRIX_SIZE / 2) *
                                          (MATRIX_SIZE / 2) * sizeof(float_t));
  float_t *lower_right = (float_t *)malloc((MATRIX_SIZE / 2) *
                                           (MATRIX_SIZE / 2) * sizeof(float_t));

  float_t *d_h_half_1, *d_h_half_2, *d_v_half_1, *d_v_half_2, *d_upper_left,
      *d_upper_right, *d_lower_left, *d_lower_right;

  checkCudaErrors(hipMalloc(&d_h_half_1, (MATRIX_SIZE * MATRIX_SIZE) / 2 *
                                              sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_h_half_2, (MATRIX_SIZE * MATRIX_SIZE) / 2 *
                                              sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_v_half_1, (MATRIX_SIZE * MATRIX_SIZE) / 2 *
                                              sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_v_half_2, (MATRIX_SIZE * MATRIX_SIZE) / 2 *
                                              sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_upper_left, (MATRIX_SIZE * MATRIX_SIZE) / 4 *
                                                sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_upper_right, (MATRIX_SIZE * MATRIX_SIZE) / 4 *
                                                 sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_lower_left, (MATRIX_SIZE * MATRIX_SIZE) / 4 *
                                                sizeof(float_t)));
  checkCudaErrors(hipMalloc(&d_lower_right, (MATRIX_SIZE * MATRIX_SIZE) / 4 *
                                                 sizeof(float_t)));

  dim3 grid(ceil(MATRIX_SIZE / float_t(TILE_WIDTH)),
            ceil(MATRIX_SIZE / float_t(TILE_WIDTH)));
  dim3 block(TILE_WIDTH, TILE_WIDTH);

  hipStream_t *streams =
      (hipStream_t *)malloc(STREAM_COUNT * sizeof(hipStream_t));
  for (unsigned int i = 0; i < STREAM_COUNT; i++) {
    hipStreamCreate(&streams[i]);
  }

  checkCudaErrors(hipMemcpyAsync(
      d_h_half_1, h_half_1, (MATRIX_SIZE * MATRIX_SIZE) / 2 * sizeof(float_t),
      hipMemcpyHostToDevice, streams[0]));
  checkCudaErrors(hipMemcpyAsync(
      d_v_half_1, v_half_1, (MATRIX_SIZE * MATRIX_SIZE) / 2 * sizeof(float_t),
      hipMemcpyHostToDevice, streams[0]));
  checkCudaErrors(hipMemcpyAsync(
      d_h_half_2, h_half_2, (MATRIX_SIZE * MATRIX_SIZE) / 2 * sizeof(float_t),
      hipMemcpyHostToDevice, streams[1]));
  checkCudaErrors(hipMemcpyAsync(
      d_v_half_2, v_half_2, (MATRIX_SIZE * MATRIX_SIZE) / 2 * sizeof(float_t),
      hipMemcpyHostToDevice, streams[1]));

  struct timespec start, end;

  clock_gettime(CLOCK_MONOTONIC, &start);

  _multiply_matrices_gpu<<<grid, block>>>(d_upper_left, d_h_half_1, d_v_half_1,
                                          MATRIX_SIZE / 2, MATRIX_SIZE / 2,
                                          MATRIX_SIZE);
  _multiply_matrices_gpu<<<grid, block>>>(d_lower_right, d_h_half_2, d_v_half_2,
                                          MATRIX_SIZE / 2, MATRIX_SIZE / 2,
                                          MATRIX_SIZE);
  _multiply_matrices_gpu<<<grid, block>>>(d_upper_right, d_h_half_1, d_v_half_2,
                                          MATRIX_SIZE / 2, MATRIX_SIZE / 2,
                                          MATRIX_SIZE);
  _multiply_matrices_gpu<<<grid, block>>>(d_lower_left, d_h_half_2, d_v_half_1,
                                          MATRIX_SIZE / 2, MATRIX_SIZE / 2,
                                          MATRIX_SIZE);

  // Wait for all threads to finish.
  checkCudaErrors(hipDeviceSynchronize());

  clock_gettime(CLOCK_MONOTONIC, &end);

  checkCudaErrors(
      hipMemcpyAsync(upper_left, d_upper_left,
                      (MATRIX_SIZE * MATRIX_SIZE) / 4 * sizeof(float_t),
                      hipMemcpyDeviceToHost, streams[0]));
  checkCudaErrors(
      hipMemcpyAsync(lower_right, d_lower_right,
                      (MATRIX_SIZE * MATRIX_SIZE) / 4 * sizeof(float_t),
                      hipMemcpyDeviceToHost, streams[0]));

  checkCudaErrors(
      hipMemcpyAsync(upper_right, d_upper_right,
                      (MATRIX_SIZE * MATRIX_SIZE) / 4 * sizeof(float_t),
                      hipMemcpyDeviceToHost, streams[0]));
  checkCudaErrors(
      hipMemcpyAsync(lower_left, d_lower_left,
                      (MATRIX_SIZE * MATRIX_SIZE) / 4 * sizeof(float_t),
                      hipMemcpyDeviceToHost, streams[0]));

  for (uint32_t i = 0; i < (MATRIX_SIZE / 2); i++) {
    memcpy(res + (i * MATRIX_SIZE), upper_left + (i * (MATRIX_SIZE / 2)),
           (MATRIX_SIZE / 2) * sizeof(float_t));
    memcpy(res + (MATRIX_SIZE / 2) + (i * MATRIX_SIZE),
           upper_right + (i * (MATRIX_SIZE / 2)),
           (MATRIX_SIZE / 2) * sizeof(float_t));
    memcpy(res + (MATRIX_SIZE * (MATRIX_SIZE / 2)) + (i * MATRIX_SIZE),
           lower_left + (i * (MATRIX_SIZE / 2)),
           (MATRIX_SIZE / 2) * sizeof(float_t));
    memcpy(res + (MATRIX_SIZE * (MATRIX_SIZE / 2)) + (MATRIX_SIZE / 2) +
               (i * MATRIX_SIZE),
           lower_right + (i * (MATRIX_SIZE / 2)),
           (MATRIX_SIZE / 2) * sizeof(float_t));
  }

  checkCudaErrors(hipFree(d_h_half_1));
  checkCudaErrors(hipFree(d_h_half_2));
  checkCudaErrors(hipFree(d_v_half_1));
  checkCudaErrors(hipFree(d_v_half_2));
  checkCudaErrors(hipFree(d_upper_left));
  checkCudaErrors(hipFree(d_upper_right));
  checkCudaErrors(hipFree(d_lower_left));
  checkCudaErrors(hipFree(d_lower_right));

  free(h_half_1);
  free(h_half_2);
  free(v_half_1);
  free(v_half_2);
  free(upper_left);
  free(upper_right);
  free(lower_left);
  free(lower_right);

  return get_time_diff_ns(start, end);
}

/*
multiply_matrices_gpu performs matrix multiplication on `m1` and `m2` using
the local tiled matrix multiplication algorithm and stores the result in
`res`.

It does not use multiple streams.

It also performs CUDA memory management and kernel invocation, using the
specified `block_width` and `block_height` for the kernel, as well as timing
the operation.

It returns the time taken to nanosecond-precision.
*/
int64_t multiply_matrices_gpu(float_t *res, float_t *m1, float_t *m2,
                              uint32_t result_width, uint32_t result_height,
                              uint32_t shared_dim) {
  float_t *d_res, *d_m1, *d_m2;

  checkCudaErrors(
      hipMalloc(&d_res, result_width * result_height * sizeof(float_t)));
  checkCudaErrors(
      hipMalloc(&d_m1, shared_dim * result_height * sizeof(float_t)));
  checkCudaErrors(
      hipMalloc(&d_m2, result_width * shared_dim * sizeof(float_t)));

  checkCudaErrors(hipMemcpy(d_m1, m1,
                             shared_dim * result_height * sizeof(float_t),
                             hipMemcpyHostToDevice));

  checkCudaErrors(hipMemcpy(d_m2, m2,
                             result_width * shared_dim * sizeof(float_t),
                             hipMemcpyHostToDevice));

  dim3 block(TILE_WIDTH, TILE_WIDTH);
  dim3 grid((result_width + TILE_WIDTH - 1) / TILE_WIDTH,
            (result_height + TILE_WIDTH - 1) / TILE_WIDTH);

  struct timespec start, end;

  clock_gettime(CLOCK_MONOTONIC, &start);

  _multiply_matrices_gpu<<<grid, block>>>(d_res, d_m1, d_m2, result_width,
                                          result_height, shared_dim);

  // Wait for all threads to finish.
  checkCudaErrors(hipDeviceSynchronize());

  clock_gettime(CLOCK_MONOTONIC, &end);

  checkCudaErrors(hipMemcpy(res, d_res,
                             result_width * result_height * sizeof(float_t),
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_res));
  checkCudaErrors(hipFree(d_m1));
  checkCudaErrors(hipFree(d_m2));

  return get_time_diff_ns(start, end);
}

// <--- MAIN FUNCTION --->

int main(void) {
  // Seed the system RNG
  srand(time(NULL));

  uint32_t m_size[2] = {MATRIX_SIZE, MATRIX_SIZE};
  uint32_t n_size[2] = {MATRIX_SIZE, MATRIX_SIZE};
  uint32_t result_size[2] = {m_size[0], n_size[1]};

  int64_t cpu_time, gpu_time, gpu_time_multistream;

  float_t *m = (float_t *)malloc(m_size[0] * m_size[1] * sizeof(float_t));
  float_t *n = (float_t *)malloc(n_size[0] * n_size[1] * sizeof(float_t));
  float_t *p_cpu =
      (float_t *)malloc(result_size[0] * result_size[1] * sizeof(float_t));
  float_t *p_gpu =
      (float_t *)malloc(result_size[0] * result_size[1] * sizeof(float_t));
  float_t *p_gpu_multistream =
      (float_t *)malloc(result_size[0] * result_size[1] * sizeof(float_t));

  if (!m || !n || !p_cpu || !p_gpu || !p_gpu_multistream) {
    fprintf(stderr, "Failed to allocate memory.\n");
    goto ERR;
  }

  printf("Tile width: %d\n", TILE_WIDTH);

  populate_matrix(m, m_size[0], m_size[1]);
  populate_matrix(n, n_size[0], n_size[1]);

  printf("Timings:\n");

  cpu_time = multiply_matrices_cpu(p_cpu, m, n, result_size[0], result_size[1],
                                   m_size[1]);

  printf("\tCPU time: %ld ns\n", cpu_time);

  gpu_time = multiply_matrices_gpu(p_gpu, n, m, result_size[0], result_size[1],
                                   m_size[1]);

  printf("\tGPU time: %ld ns\n", gpu_time);

  if (!compare_outputs(p_cpu, p_gpu, EPSILON, result_size[0], result_size[1]))
    goto ERR;

  gpu_time_multistream =
      multiply_matrices_gpu_multistream(p_gpu_multistream, m, n);

  printf("\tGPU time (multistream): %ld ns\n", gpu_time_multistream);

  if (!compare_outputs(p_gpu, p_gpu_multistream, EPSILON, result_size[0],
                       result_size[1]))
    goto ERR;

ERR:
  free(m);
  free(n);
  free(p_cpu);
  free(p_gpu);
  free(p_gpu_multistream);

  return 0;
}
