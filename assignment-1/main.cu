
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main(void) {
    printf("Hello World from CPU!\n");
    helloFromGPU<<<1, 10>>>();
    hipDeviceSynchronize();
    return 0;
}
